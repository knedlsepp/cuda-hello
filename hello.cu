

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int n) 
{ 	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
		c[index] = a[index] + b[index];
}

void fill(int *a, int len) {
	while(len--) {
		a[len] = len;
		//printf("file m[%d] = %d\n", len, len);
	}
}

#define N 16
int main(void) 
{ 		
	int *a, *b, *c;       // host copies of a, b, c
	int *a1, *b1, *c1; // device copies of a, b, c
	int nsize = N * sizeof(int);

	hipMalloc((void **)&a1, nsize); // allocate space for device
	hipMalloc((void **)&b1, nsize);
	hipMalloc((void **)&c1, nsize);
	a = (int *)malloc(nsize); // fill-in a with values
	b = (int *)malloc(nsize); // fill-in b with values
	c = (int *)malloc(nsize);
	fill(a, N);
	fill(b, N);
	hipMemcpy(a1, a, nsize, hipMemcpyHostToDevice);
	hipMemcpy(b1, b, nsize, hipMemcpyHostToDevice);

	// format: add<<< BLOCKS, THREADS >>>  access via blockIdx.x and threadIdx.x
	add<<<N,1>>>(a1, b1, c1, N); // seen as N/THREADS_PER_BLOCK,THREADS_PER_BLOCK or (N + M-1) / M,M	
	hipMemcpy(c, c1, nsize, hipMemcpyDeviceToHost); // result back to host

	for(int x = 0; x < N; x++) {
		printf("c[%d] = %d a[%d] + b[%d] = (%d + %d)\n", x, c[x], x, x, a[x], b[x]);
	}
	hipFree(a1); hipFree(b1); hipFree(c1); free(a); free(b); free(c);
	return 0; 	
}
